
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdarg.h>
#include <list>
#include <math.h>
#include <hip/hip_vector_types.h>//allows the use of composite types, e.g. float2, float3, int3, etc.

#define BLOCKSIZE 32

const float PI = 3.141592653589793238462643383279502884;

const char *filepath = "data.txt";

int xSize = 512;
int ySize = 512;
__device__ int leftMargin = 64;
__device__ int rightMargin = 448;
__device__ int bottomMargin = 64;
__device__ int topMargin = 448;

// How hard the boid can turn to avoid walls
__device__ float turnFactor = 0.2;
// The distance within which separation occurs
float avoidRange = 8;
__device__ float sqrAvoidRange = 64;
// The rate at which separation occurs
__device__ float avoidFactor = 0.15;
// The distance within which alignment occurs
float visualRange = 20;
__device__ float sqrVisualRange = 400;
// The rate at which alignment occurs
__device__ float matchingFactor = 0.05;
// The rate at which cohesion occurs
__device__ float cohesionFactor = 0.2;
// The minimum speed of the boids
__device__ float minSpeed = 1;
// The maximum speed of the boids
__device__ float maxSpeed = 2;
// The formation angle
__device__ float formationAngle = 0.7 * PI;

// Calculate cell size
//const int cellSize = pow(2, ceil(log2(std::max(avoidRange, visualRange))));
// I failed to implement this using constexpr so you (the user) must compute manually
// const int numCells_x = xSize / cellSize;
// const int numCells_y = ySize / cellSize;
// const int numCells_x = 16;
// const int numCells_y = 16;






void setVars(int _xSize, int _ySize, int _marginSize, 
        float _turnFactor, 
        float _avoidRange, float _avoidFactor,
        float _visualRange, float _matchingFactor,
        float _cohesionFactor,
        float _minSpeed, float _maxSpeed,
        float _formationAngle)
{
    xSize = _xSize;
    ySize = _ySize;

    int _leftMargin = _marginSize;
    hipMemcpyToSymbol(HIP_SYMBOL(leftMargin), &_leftMargin, sizeof(int), 0, hipMemcpyHostToDevice);
    int _rightMargin = _xSize - _marginSize;
    hipMemcpyToSymbol(HIP_SYMBOL(rightMargin), &_rightMargin, sizeof(int), 0, hipMemcpyHostToDevice);
    int _bottomMargin = _marginSize;
    hipMemcpyToSymbol(HIP_SYMBOL(bottomMargin), &_bottomMargin, sizeof(int), 0, hipMemcpyHostToDevice);
    int _topMargin = _ySize - _marginSize;
    hipMemcpyToSymbol(HIP_SYMBOL(topMargin), &_topMargin, sizeof(int), 0, hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(turnFactor), &_turnFactor, sizeof(float), 0, hipMemcpyHostToDevice);

    float _sqrAvoidRange = _avoidRange * _avoidRange;
    hipMemcpyToSymbol(HIP_SYMBOL(sqrAvoidRange), &_sqrAvoidRange, sizeof(float), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(avoidFactor), &_avoidFactor, sizeof(float), 0, hipMemcpyHostToDevice);

    float _sqrVisualRange = _visualRange * _visualRange;
    hipMemcpyToSymbol(HIP_SYMBOL(sqrVisualRange), &_sqrVisualRange, sizeof(float), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(matchingFactor), &_matchingFactor, sizeof(float), 0, hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(cohesionFactor), &_cohesionFactor, sizeof(float), 0, hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(minSpeed), &_minSpeed, sizeof(float), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(maxSpeed), &_maxSpeed, sizeof(float), 0, hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(formationAngle), &_formationAngle, sizeof(float), 0, hipMemcpyHostToDevice);    
}



/*
Boid class to represent a single bird / particle / actor
*/
class Boid {
    public:
        float px;
        float py;
        float vx;
        float vy;

        Boid() {
            this->px = 0;
            this->py = 0;
            this->vx = 0;
            this->vy = 0;
        }
    
        Boid(float px, float py, float vx, float vy) {
            this->px = px;
            this->py = py;
            this->vx = vx;
            this->vy = vy;
        }
};

Boid* boidsArray;



/*
Cell class to represent a subdivision of the simulation space containing a list of pointers to boids
*/
struct Cell {
    std::list<Boid*> boids;
};



/*
Saves 
*/
void save(FILE *fptr, int numBoids, int frameNumber) {
    // Write vector array to file
    fprintf(fptr, "Frame %d\n", frameNumber);
    for(int i = 0; i < numBoids; i++) {
        fprintf(fptr, "%f %f %f %f\n", boidsArray[i].px, boidsArray[i].py, boidsArray[i].vx, boidsArray[i].vy);
    }    
}



// Sourced from https://stackoverflow.com/questions/686353/random-float-number-generation
float randFloat(float min, float max) {
    return min + static_cast <float> (rand()) /( static_cast <float> (RAND_MAX/(max-min)));
}



/*
Magnitude of a vector
*/
__device__ float mag(float x, float y) {
    return sqrt(x * x + y * y);
}



/*
Square magnitude of a vector
*/
__device__ float sqrMag(float x, float y) {
    return x * x + y * y;
}



/*
Vector dot product
*/
__device__ float dot(float x1, float y1, float x2, float y2) {
    return x1 * x2 + y1 * y2;
}



/*
Gets the orthogonal vector using pass-by-reference
*/
__device__ void getOrthogonal(float &orthogonalVector_x, float &orthogonalVector_y, 
        float diffVector_x, float diffVector_y, 
        float formationDir_x, float formationDir_y) {
    //TODO expand and simplify    
    // Determines whether formation is to the left or right of boid using cross product
    // And constructs formation vector accordingly by rotating formationDir
    float formationVector_x, formationVector_y;
    float c = cos(formationAngle);
    float s = sin(formationAngle);
    if (formationDir_x * diffVector_y - formationDir_y * diffVector_x > 0) { //If boid is to the left of formation
        formationVector_x = formationDir_x * c - formationDir_y * s;
        formationVector_y = formationDir_x * s - formationDir_y * c;
    }
    else { //If boid is to the right of formation
        formationVector_x = formationDir_x * c + formationDir_y * s;
        formationVector_y = formationDir_y * c - formationDir_x * s;
    }

    // Check that formation is ahead of boid not behind (the dot product is more than 90 degrees)
    // This is to stop the leaders from trying to fall in line behind
    if (dot(formationDir_x, formationDir_y, diffVector_x, diffVector_y) < 0) {
        // Get at the vector orthogonal to the formationVector and move in that direction
        float sqrM = sqrMag(formationVector_x, formationVector_y);
        float val = dot(diffVector_x, diffVector_y, formationVector_x, formationVector_y) / sqrM;
        orthogonalVector_x = val * formationVector_x - diffVector_x;
        orthogonalVector_y = val * formationVector_y - diffVector_y;
    }
}



//Using shared memory
__global__ void updateBoidsKernel_GPU(int N, const Boid* in, Boid* out)
{
    //This boid's index
    //for 1D
    int thisIndex = blockIdx.x * (blockDim.x * blockDim.y) + threadIdx.y * blockDim.x + threadIdx.x; 

    //printf("kernel: %f %f\n", out[0].px, out[0].py);
    //printf("%f %f %f %f\n", in[thisIndex].px, in[thisIndex].py, in[thisIndex].vx, in[thisIndex].vy);
    //for 2D
    // int x = blockIdx.x * blockDim.x + threadIdx.x;
    // int y = blockIdx.y * blockDim.y + threadIdx.y;
    // int thisIndex = x + (blockDim.x * gridDim.x) * y;


    if (thisIndex < N) { //Check out of bounds
        float avoidVector_x = 0, avoidVector_y = 0;
        float formationDir_x = 0, formationDir_y = 0;
        float formationPos_x = 0, formationPos_y = 0;
        int neighboringBoids = 0;

        Boid b = in[thisIndex];

        for (int otherIndex = 0; otherIndex < N; otherIndex++) {
            if (otherIndex == thisIndex) continue; //Ignore itself

            Boid o = in[otherIndex];

            // Get the distance between this boid and other boid
            float sqrDist = sqrMag(b.px - o.px, b.py - o.py);
            if (sqrDist < sqrAvoidRange) { // If the distance is less than protected range
                //Divide by the square of distance to make avoidance exponential and smoother
                avoidVector_x += (b.px - o.px) / sqrDist;
                avoidVector_y += (b.py - o.py) / sqrDist;
            }
            if (sqrDist < sqrVisualRange) { // If the distance is less than visual range
                formationDir_x += o.vx;
                formationDir_y += o.vy;
                formationPos_x += o.px;
                formationPos_y += o.py;
                neighboringBoids++;
            }

        }

        //Make a copy for out
        //TODO make a pointer named bOut
        out[thisIndex] = b;

        // Separation - move away from nearby boids
        out[thisIndex].vx += avoidVector_x * avoidFactor;
        out[thisIndex].vy += avoidVector_y * avoidFactor;

        if (neighboringBoids > 0) { //If there were any boids in visual range
            // Get mean formation direction and position
            formationDir_x /= neighboringBoids;
            formationDir_y /= neighboringBoids;
            formationPos_x /= neighboringBoids;
            formationPos_y /= neighboringBoids;  

            // Alignment - match the mean velocity of all boids in visual range
            out[thisIndex].vx += (formationDir_x - out[thisIndex].vx) * matchingFactor;
            out[thisIndex].vy += (formationDir_y - out[thisIndex].vy) * matchingFactor;

            // Flocking
            // Represents a vector pointed dowards this boid from the centre of mass
            float diffVector_x = out[thisIndex].px - formationPos_x;
            float diffVector_y = out[thisIndex].py - formationPos_y;

            float orthogonalVector_x = 0, orthogonalVector_y = 0;
            getOrthogonal(orthogonalVector_x, orthogonalVector_y, 
                    diffVector_x, diffVector_y,
                    formationDir_x, formationDir_y);

            out[thisIndex].vx += orthogonalVector_x * cohesionFactor;
            out[thisIndex].vy += orthogonalVector_y * cohesionFactor;            
        }



        // Avoid edges
        if (out[thisIndex].px < leftMargin) {
            out[thisIndex].vx += turnFactor;
        }
        else if (out[thisIndex].px > rightMargin) {
            out[thisIndex].vx -= turnFactor;
        }
        if (out[thisIndex].py < bottomMargin) {
            out[thisIndex].vy += turnFactor;
        }
        else if (out[thisIndex].py > topMargin) {
            out[thisIndex].vy -= turnFactor;
        }
        //---------------------------------------



        // Impose speed limit on boid
        //TODO use *= and precalculate maxSpeed / speed
        float speed = mag(out[thisIndex].vx, out[thisIndex].vy);
        if (speed > maxSpeed) {
            out[thisIndex].vx *= maxSpeed / speed;
            out[thisIndex].vy *= maxSpeed / speed;
        }
        else if (speed == 0) {
            // TODO
        }
        else if (speed < minSpeed) {
            out[thisIndex].vx *= minSpeed / speed;
            out[thisIndex].vy *= minSpeed / speed;
        }
        //---------------------------------------

        
        
        // Update boid position
        out[thisIndex].px += out[thisIndex].vx;
        out[thisIndex].py += out[thisIndex].vy;
        //---------------------------------------
    }
}



__host__ void updateBoids_GPU(int N, const Boid* in, Boid* out)
{
    //TODO is it possible to use a single pointer??

    size_t size = N * sizeof(Boid);

    //Allocate memory on the device
    Boid *deviceIn;
    Boid *deviceOut;
    hipMalloc(&deviceIn, size);
    hipMalloc(&deviceOut, size);

    //Copy memory from host to device
    hipMemcpy(deviceIn, in, size, hipMemcpyHostToDevice);
    //cudaMemcpy(deviceOut, out, size, cudaMemcpyHostToDevice);

    //Specify blocks and threads
    dim3 threads(BLOCKSIZE, BLOCKSIZE);
    dim3 blocks((N + BLOCKSIZE - 1) / BLOCKSIZE, 1); //Ceil division of N / BLOCKSIZE

    //Run
    printf("running kernel\n");
    updateBoidsKernel_GPU<<<blocks, threads>>>(N, deviceIn, deviceOut);

    //Copy memory from device to host
    hipMemcpy(out, deviceOut, size, hipMemcpyDeviceToHost);

    //Clean up
    hipFree(deviceIn);
    hipFree(deviceOut);
}



void init(int numBoids)
{
    boidsArray = (Boid*)malloc(numBoids * sizeof(Boid));

    // Initialise array of boids and assign them to cells
    for(int i = 0; i < numBoids; i++) {
        float px = randFloat(0, xSize);
        float py = randFloat(0, ySize);
        // Random normalised direction
        float randTheta = randFloat(0, 2 * PI);
        float vx = cos(randTheta) * 1.0;
        float vy = sin(randTheta) * 1.0;
        boidsArray[i] = Boid(px, py, vx, vy);
    }
}



int main()
{
    int numBoids = 1000;
    int numFrames = 300;


    // Create a file and open it for writing
    FILE *fptr;
    fptr = fopen(filepath, "w");
    if (fptr == NULL) {
        printf("%s", "Error opening file");
        return 1;
    }


    //Variables for main
    setVars(512, 512, 64, 0.2, 8, 0.15, 20, 0.05, 0.2, 1, 2, 0.7 * PI);

    init(numBoids);
    printf("Init complete\n");

    save(fptr, numBoids, 0);



    // Update boids
    for (int frame = 1; frame < numFrames; frame++) {
        Boid* out = (Boid*)malloc(numBoids * sizeof(Boid));
        updateBoids_GPU(numBoids, boidsArray, out);
        memcpy(boidsArray, out, sizeof(boidsArray));
        printf("Frame %d complete\n", frame);
        
        save(fptr, numBoids, frame);
    }


    
    // Close the file
    fclose(fptr);



    return 0;
}