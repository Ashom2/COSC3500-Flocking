#include "hip/hip_runtime.h"
#include "flockingGPU.cuh"

#define BLOCKSIZE 512

const float PI = 3.141592653589793238462643383279502884;

const char *filepath = "data.txt";

int xSize = 512;
int ySize = 512;
__device__ int leftMargin = 64;
__device__ int rightMargin = 448;
__device__ int bottomMargin = 64;
__device__ int topMargin = 448;

// How hard the boid can turn to avoid walls
__device__ float turnFactor = 0.2;
// The distance within which separation occurs
float avoidRange = 8;
__device__ float sqrAvoidRange = 64;
// The rate at which separation occurs
__device__ float avoidFactor = 0.15;
// The distance within which alignment occurs
float visualRange = 20;
__device__ float sqrVisualRange = 400;
// The rate at which alignment occurs
__device__ float matchingFactor = 0.05;
// The rate at which cohesion occurs
__device__ float cohesionFactor = 0.2;
// The minimum speed of the boids
__device__ float minSpeed = 1;
// The maximum speed of the boids
__device__ float maxSpeed = 2;
// The formation angle
__device__ float formationAngle = 0.7 * PI;

// Calculate cell size
float cellSize = 32;
int numCells_x = 16;
int numCells_y = 16;
__device__ int deviceNumCells_x = 16;
__device__ int deviceNumCells_y = 16;
//const int cellSize = pow(2, ceil(log2(std::max(avoidRange, visualRange))));
// I failed to implement this using constexpr so you (the user) must compute manually







void setVars(int _xSize, int _ySize, int _marginSize, 
        float _turnFactor, 
        float _avoidRange, float _avoidFactor,
        float _visualRange, float _matchingFactor,
        float _cohesionFactor,
        float _minSpeed, float _maxSpeed,
        float _formationAngle)
{
    xSize = _xSize;
    ySize = _ySize;

    int _leftMargin = _marginSize;
    hipMemcpyToSymbol(HIP_SYMBOL(leftMargin), &_leftMargin, sizeof(int), 0, hipMemcpyHostToDevice);
    int _rightMargin = _xSize - _marginSize;
    hipMemcpyToSymbol(HIP_SYMBOL(rightMargin), &_rightMargin, sizeof(int), 0, hipMemcpyHostToDevice);
    int _bottomMargin = _marginSize;
    hipMemcpyToSymbol(HIP_SYMBOL(bottomMargin), &_bottomMargin, sizeof(int), 0, hipMemcpyHostToDevice);
    int _topMargin = _ySize - _marginSize;
    hipMemcpyToSymbol(HIP_SYMBOL(topMargin), &_topMargin, sizeof(int), 0, hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(turnFactor), &_turnFactor, sizeof(float), 0, hipMemcpyHostToDevice);

    float _sqrAvoidRange = _avoidRange * _avoidRange;
    hipMemcpyToSymbol(HIP_SYMBOL(sqrAvoidRange), &_sqrAvoidRange, sizeof(float), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(avoidFactor), &_avoidFactor, sizeof(float), 0, hipMemcpyHostToDevice);

    float _sqrVisualRange = _visualRange * _visualRange;
    hipMemcpyToSymbol(HIP_SYMBOL(sqrVisualRange), &_sqrVisualRange, sizeof(float), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(matchingFactor), &_matchingFactor, sizeof(float), 0, hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(cohesionFactor), &_cohesionFactor, sizeof(float), 0, hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(minSpeed), &_minSpeed, sizeof(float), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(maxSpeed), &_maxSpeed, sizeof(float), 0, hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(formationAngle), &_formationAngle, sizeof(float), 0, hipMemcpyHostToDevice);    
}



/*
Retrieves the x index of a cell at position x
*/
int getCell_x(float x) {
    int nx = (int)(x / cellSize);
    //clamp
    if (nx > numCells_x - 1) return numCells_x - 1;
    if (nx < 0) return 0;
    return nx;
}

/*
Retrieves the y index of a cell at position y
*/
int getCell_y(float y) {
    int ny = (int)(y / cellSize);
    //clamp
    if (ny > numCells_y - 1) return numCells_y - 1;
    if (ny < 0) return 0;
    return ny;
}

/*
Retrieves the 1D index of a cell at position x and y
*/
int getCell_i(float x, float y) {
    return getCell_x(x) + getCell_y(y) * numCells_x;
}



/*
Boid class to represent a single bird / particle / actor
*/
class Boid {
    public:
        float px;
        float py;
        float vx;
        float vy;
        uint cellIndex;
    
        Boid(float px, float py, float vx, float vy) {
            this->px = px;
            this->py = py;
            this->vx = vx;
            this->vy = vy;
            this->cellIndex = getCell_i(px, py);
        }
};

Boid* boidsArrayHost;
Boid** cellsArrayHost;
uint* cellOffsetsHost;
uint* cellSizesHost;



void freeMemory() {
    free(boidsArrayHost);
    for(int i = 0; i < numCells_x * numCells_y; i++) free(cellsArrayHost[i]);
    free(cellsArrayHost);
    free(cellOffsetsHost);
    free(cellSizesHost);
}

/*
Saves 
*/
void save(FILE *fptr, int numBoids, int frameNumber) {
    // Write vector array to file
    fprintf(fptr, "Frame %d\n", frameNumber);
    for(int i = 0; i < numBoids; i++) {
        fprintf(fptr, "%f %f %f %f\n", boidsArrayHost[i].px, boidsArrayHost[i].py, boidsArrayHost[i].vx, boidsArrayHost[i].vy);
    }    
}



// Sourced from https://stackoverflow.com/questions/686353/random-float-number-generation
float randFloat(float min, float max) {
    return min + static_cast <float> (rand()) /( static_cast <float> (RAND_MAX/(max-min)));
}



/*
Magnitude of a vector
*/
__device__ float mag(float x, float y) {
    return sqrt(x * x + y * y);
}



/*
Square magnitude of a vector
*/
__device__ float sqrMag(float x, float y) {
    return x * x + y * y;
}



/*
Vector dot product
*/
__device__ float dot(float x1, float y1, float x2, float y2) {
    return x1 * x2 + y1 * y2;
}



/*
Gets the orthogonal vector using pass-by-reference
*/
__device__ void getOrthogonal(float &orthogonalVector_x, float &orthogonalVector_y, 
        float diffVector_x, float diffVector_y, 
        float formationDir_x, float formationDir_y) {
    //TODO expand and simplify    
    // Determines whether formation is to the left or right of boid using cross product
    // And constructs formation vector accordingly by rotating formationDir
    float formationVector_x, formationVector_y;
    float c = cos(formationAngle);
    float s = sin(formationAngle);
    if (formationDir_x * diffVector_y - formationDir_y * diffVector_x > 0) { //If boid is to the left of formation
        formationVector_x = formationDir_x * c - formationDir_y * s;
        formationVector_y = formationDir_x * s - formationDir_y * c;
    }
    else { //If boid is to the right of formation
        formationVector_x = formationDir_x * c + formationDir_y * s;
        formationVector_y = formationDir_y * c - formationDir_x * s;
    }

    // Check that formation is ahead of boid not behind (the dot product is more than 90 degrees)
    // This is to stop the leaders from trying to fall in line behind
    if (dot(formationDir_x, formationDir_y, diffVector_x, diffVector_y) < 0) {
        // Get at the vector orthogonal to the formationVector and move in that direction
        float sqrM = sqrMag(formationVector_x, formationVector_y);
        float val = dot(diffVector_x, diffVector_y, formationVector_x, formationVector_y) / sqrM;
        orthogonalVector_x = val * formationVector_x - diffVector_x;
        orthogonalVector_y = val * formationVector_y - diffVector_y;
    }
}



//Using shared memory
__global__ void updateBoidsKernel_GPU(int N, Boid* cellsArray, uint* cellOffsetsArray, uint* cellSizesArray, Boid* out)
{
    //This boid's index
    int thisIndex = blockIdx.x * blockDim.x + threadIdx.x;

    if (thisIndex < N) { //Check out of bounds
        float avoidVector_x = 0, avoidVector_y = 0;
        float formationDir_x = 0, formationDir_y = 0;
        float formationPos_x = 0, formationPos_y = 0;
        int neighboringBoids = 0;

        //For some reason Boid& o = cellsArray[otherIndex]; seems to make it faster but making this line a reference makes it slower
        Boid b = cellsArray[thisIndex];

        //TODO there must be a more efficient way of doing this
        int thisCellIndex = b.cellIndex;
        int cell_x = thisCellIndex % deviceNumCells_x;
        int cell_y = thisCellIndex / deviceNumCells_x;
        // Iterate over neighboring cells
        for(int x = cell_x - 1; x <= cell_x + 1; x++) {
            if (x < 0 || x >= deviceNumCells_x) continue; //Ignore cells beyond boundary
            for(int y = cell_y - 1; y <= cell_y + 1; y++) {
                if (y < 0 || y >= deviceNumCells_y) continue; //Ignore cells beyond boundary

                uint otherCellIndex = x + y * deviceNumCells_x;

                //Iterate over each boid in neighboring cell
                for (int otherIndex = cellOffsetsArray[otherCellIndex]; otherIndex < cellOffsetsArray[otherCellIndex] + cellSizesArray[otherCellIndex]; otherIndex++) {

                    Boid& o = cellsArray[otherIndex];

                    if (&o == &cellsArray[thisIndex]) continue; //Ignore itself

                    //printf("          This cell (%d, %d): Found boid (%.0f, %.0f). in cell: %d, %d\n", cell_x, cell_y, o.px, o.py, x, y);

                    // Get the distance between this boid and other boid
                    float sqrDist = sqrMag(b.px - o.px, b.py - o.py);
                    if (sqrDist < sqrAvoidRange) { // If the distance is less than protected range
                        //Divide by the square of distance to make avoidance exponential and smoother
                        avoidVector_x += (b.px - o.px) / sqrDist;
                        avoidVector_y += (b.py - o.py) / sqrDist;
                    }
                    if (sqrDist < sqrVisualRange) { // If the distance is less than visual range
                        formationDir_x += o.vx;
                        formationDir_y += o.vy;
                        formationPos_x += o.px;
                        formationPos_y += o.py;
                        neighboringBoids++;
                    }
                }
            }
        }

        //Make a copy for out
        Boid& bOut = out[thisIndex]; //Reference to out[thisIndex]
        bOut = b;

        // Separation - move away from nearby boids
        bOut.vx += avoidVector_x * avoidFactor;
        bOut.vy += avoidVector_y * avoidFactor;

        if (neighboringBoids > 0) { //If there were any boids in visual range
            // Get mean formation direction and position
            formationDir_x /= neighboringBoids;
            formationDir_y /= neighboringBoids;
            formationPos_x /= neighboringBoids;
            formationPos_y /= neighboringBoids;  

            // Alignment - match the mean velocity of all boids in visual range
            bOut.vx += (formationDir_x - bOut.vx) * matchingFactor;
            bOut.vy += (formationDir_y - bOut.vy) * matchingFactor;

            // Flocking
            // Represents a vector pointed dowards this boid from the centre of mass
            float diffVector_x = bOut.px - formationPos_x;
            float diffVector_y = bOut.py - formationPos_y;

            float orthogonalVector_x = 0, orthogonalVector_y = 0;
            getOrthogonal(orthogonalVector_x, orthogonalVector_y, 
                    diffVector_x, diffVector_y,
                    formationDir_x, formationDir_y);

            bOut.vx += orthogonalVector_x * cohesionFactor;
            bOut.vy += orthogonalVector_y * cohesionFactor;            
        }



        // Avoid edges
        if (bOut.px < leftMargin) {
            bOut.vx += turnFactor;
        }
        else if (bOut.px > rightMargin) {
            bOut.vx -= turnFactor;
        }
        if (bOut.py < bottomMargin) {
            bOut.vy += turnFactor;
        }
        else if (bOut.py > topMargin) {
            bOut.vy -= turnFactor;
        }
        //---------------------------------------



        // Impose speed limit on boid
        float speed = mag(bOut.vx, bOut.vy);
        if (speed > maxSpeed) {
            bOut.vx *= maxSpeed / speed;
            bOut.vy *= maxSpeed / speed;
        }
        else if (speed < minSpeed) {
            bOut.vx *= minSpeed / speed;
            bOut.vy *= minSpeed / speed;
        }
        //else if (speed == 0) {} // TODO
        //---------------------------------------

        
        
        // Update boid position
        bOut.px += bOut.vx;
        bOut.py += bOut.vy;
        //---------------------------------------
    }
}



void assignBoidsToCells(int numBoids)
{
    //Reset cells
    for(int i = 0; i < numCells_x * numCells_y; i++) {
        cellsArrayHost[i] = (Boid*)malloc(0);
        cellSizesHost[i] = 0;
        cellOffsetsHost[i] = 0;
    }

    //Determine cell sizes
    for(int i = 0; i < numBoids; i++) {
        Boid& b = boidsArrayHost[i];
        b.cellIndex = getCell_i(b.px, b.py); //TODO dont calculate this here
        //increase cell size
        cellSizesHost[b.cellIndex]++;
    }

    //Reallocate memory
    for(int i = 0; i < numCells_x * numCells_y; i++) {
        cellsArrayHost[i] = (Boid*)realloc(cellsArrayHost[i], cellSizesHost[i] * sizeof(Boid));
        cellSizesHost[i] = 0; //Reset to 0 so we can use as an incrementer
    }

    //Put boids in cells
    for(int i = 0; i < numBoids; i++) {
        Boid& b = boidsArrayHost[i];
        //Allocate boid to a cell in cellsArrayHost
        cellsArrayHost[b.cellIndex][cellSizesHost[b.cellIndex]++] = b;
    }
}



__host__ float updateBoids_GPU(int N)
{
    size_t size1 = N * sizeof(Boid);
    size_t size2 = numCells_x * numCells_y * sizeof(uint);
    
    auto start = std::chrono::high_resolution_clock::now();

    //Recalculate which cells have which boids
    assignBoidsToCells(N);

    

    //flatten cells array so we can put it onto the kernel
    Boid* flattenedCellsArray = (Boid*)malloc(size1);
    int indexOffset = 0;
    for (int i = 0; i < numCells_x * numCells_y; i++) {
        cellOffsetsHost[i] = indexOffset;
        for (int j = 0; j < cellSizesHost[i]; j++) {
            flattenedCellsArray[indexOffset + j] = cellsArrayHost[i][j];
        }
        indexOffset += cellSizesHost[i];
    }


    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::micro> duration_us = end - start;
    printf("N: %d. ------------- CPU Time taken: %g us.\n", N, duration_us.count());

    //Allocate memory on the device
    Boid* deviceCells;
    uint *deviceCellOffsets;
    uint *deviceCellSizes;
    Boid *deviceOut;
    hipMalloc(&deviceCells, size1);
    hipMalloc(&deviceCellOffsets, size2);
    hipMalloc(&deviceCellSizes, size2);
    hipMalloc(&deviceOut, size1);

    //Copy memory from host to device
    hipMemcpy(deviceCells, flattenedCellsArray, size1, hipMemcpyHostToDevice);
    hipMemcpy(deviceCellOffsets, cellOffsetsHost, size2, hipMemcpyHostToDevice);
    hipMemcpy(deviceCellSizes, cellSizesHost, size2, hipMemcpyHostToDevice);
    //printf("Device Variable Copying:\t%s\n", hipGetErrorString(hipGetLastError()));

    //Specify blocks and threads, using 1D threads and blocks since our data is 1D
    int threadsPerBlock = BLOCKSIZE;
    int numBlocks = (N + threadsPerBlock - 1) / threadsPerBlock; //Int division of N / threadsPerBlock

    //Start Timer
    hipEvent_t startGPU;
    hipEvent_t stopGPU;
    hipEventCreate(&startGPU);
    hipEventCreate(&stopGPU);
    hipEventRecord(startGPU);
    
    //Run
    updateBoidsKernel_GPU<<<numBlocks, threadsPerBlock>>>(N, deviceCells, deviceCellOffsets, deviceCellSizes, deviceOut);

    //Mark end of kernel execution
    hipEventRecord(stopGPU);
    //Wait for results to be available
    hipEventSynchronize(stopGPU);   
    //Measure the time the kernel took to execute
    float kernelTime;
    hipEventElapsedTime(&kernelTime, startGPU, stopGPU);

    //Wait for the kernel to synchronise
    hipDeviceSynchronize();

    //Copy memory from device to host
    hipMemcpy(boidsArrayHost, deviceOut, size1, hipMemcpyDeviceToHost);

    //Clean up
    hipFree(deviceCells);
    hipFree(deviceCellOffsets);
    hipFree(deviceCellSizes);
    hipFree(deviceOut);

    return kernelTime * 1000;
}



void init(int numBoids)
{
    //Malloc boids
    boidsArrayHost = (Boid*)malloc(numBoids * sizeof(Boid));

    //Malloc cells
    cellsArrayHost = (Boid**)malloc(numCells_x * numCells_y * sizeof(Boid*));
    cellSizesHost = (uint*)malloc(numCells_x * numCells_y * sizeof(uint));
    cellOffsetsHost = (uint*)malloc(numCells_x * numCells_y * sizeof(uint));

    // Initialise array of boids and assign them to cells
    for(int i = 0; i < numBoids; i++) {
        float px = randFloat(0, xSize);
        float py = randFloat(0, ySize);
        // Random normalised direction
        float randTheta = randFloat(0, 2 * PI);
        float vx = cos(randTheta) * 1.0;
        float vy = sin(randTheta) * 1.0;
        boidsArrayHost[i] = Boid(px, py, vx, vy);
    }
}


/*
int main()
{
    int numBoids = 1000;
    int numFrames = 300;



    // Create a file and open it for writing
    FILE *fptr;
    fptr = fopen(filepath, "w");
    if (fptr == NULL) {
        printf("%s", "Error opening file");
        return 1;
    }



    //Variables for main
    setVars(512, 512, 64, 0.2, 8, 0.15, 20, 0.05, 0.2, 1, 2, 0.7 * PI);

    init(numBoids);

    save(fptr, numBoids, 0);



    // Update boids
    for (int frame = 1; frame < numFrames; frame++) {
        //Change the state and position of boids on the GPU
        updateBoids_GPU(numBoids);
        
        save(fptr, numBoids, frame);
    }

    freeMemory();
    
    // Close the file
    fclose(fptr);



    return 0;
}
*/