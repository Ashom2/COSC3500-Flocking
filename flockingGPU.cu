#include "hip/hip_runtime.h"
#include "flockingGPU.cuh"

#define BLOCKSIZE 512

const float PI = 3.141592653589793238462643383279502884;

const char *filepath = "data.txt";

int xSize = 512;
int ySize = 512;
__device__ int leftMargin = 64;
__device__ int rightMargin = 448;
__device__ int bottomMargin = 64;
__device__ int topMargin = 448;

// How hard the boid can turn to avoid walls
__device__ float turnFactor = 0.2;
// The distance within which separation occurs
float avoidRange = 8;
__device__ float sqrAvoidRange = 64;
// The rate at which separation occurs
__device__ float avoidFactor = 0.15;
// The distance within which alignment occurs
float visualRange = 20;
__device__ float sqrVisualRange = 400;
// The rate at which alignment occurs
__device__ float matchingFactor = 0.05;
// The rate at which cohesion occurs
__device__ float cohesionFactor = 0.2;
// The minimum speed of the boids
__device__ float minSpeed = 1;
// The maximum speed of the boids
__device__ float maxSpeed = 2;
// The formation angle
__device__ float formationAngle = 0.7 * PI;

// Calculate cell size
//const int cellSize = pow(2, ceil(log2(std::max(avoidRange, visualRange))));
// I failed to implement this using constexpr so you (the user) must compute manually
// const int numCells_x = xSize / cellSize;
// const int numCells_y = ySize / cellSize;
// const int numCells_x = 16;
// const int numCells_y = 16;






void setVars(int _xSize, int _ySize, int _marginSize, 
        float _turnFactor, 
        float _avoidRange, float _avoidFactor,
        float _visualRange, float _matchingFactor,
        float _cohesionFactor,
        float _minSpeed, float _maxSpeed,
        float _formationAngle)
{
    xSize = _xSize;
    ySize = _ySize;

    int _leftMargin = _marginSize;
    hipMemcpyToSymbol(HIP_SYMBOL(leftMargin), &_leftMargin, sizeof(int), 0, hipMemcpyHostToDevice);
    int _rightMargin = _xSize - _marginSize;
    hipMemcpyToSymbol(HIP_SYMBOL(rightMargin), &_rightMargin, sizeof(int), 0, hipMemcpyHostToDevice);
    int _bottomMargin = _marginSize;
    hipMemcpyToSymbol(HIP_SYMBOL(bottomMargin), &_bottomMargin, sizeof(int), 0, hipMemcpyHostToDevice);
    int _topMargin = _ySize - _marginSize;
    hipMemcpyToSymbol(HIP_SYMBOL(topMargin), &_topMargin, sizeof(int), 0, hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(turnFactor), &_turnFactor, sizeof(float), 0, hipMemcpyHostToDevice);

    float _sqrAvoidRange = _avoidRange * _avoidRange;
    hipMemcpyToSymbol(HIP_SYMBOL(sqrAvoidRange), &_sqrAvoidRange, sizeof(float), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(avoidFactor), &_avoidFactor, sizeof(float), 0, hipMemcpyHostToDevice);

    float _sqrVisualRange = _visualRange * _visualRange;
    hipMemcpyToSymbol(HIP_SYMBOL(sqrVisualRange), &_sqrVisualRange, sizeof(float), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(matchingFactor), &_matchingFactor, sizeof(float), 0, hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(cohesionFactor), &_cohesionFactor, sizeof(float), 0, hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(minSpeed), &_minSpeed, sizeof(float), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(maxSpeed), &_maxSpeed, sizeof(float), 0, hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(formationAngle), &_formationAngle, sizeof(float), 0, hipMemcpyHostToDevice);    
}



/*
Boid class to represent a single bird / particle / actor
*/
class Boid {
    public:
        float px;
        float py;
        float vx;
        float vy;

        Boid() {
            this->px = 0;
            this->py = 0;
            this->vx = 0;
            this->vy = 0;
        }
    
        Boid(float px, float py, float vx, float vy) {
            this->px = px;
            this->py = py;
            this->vx = vx;
            this->vy = vy;
        }
};

Boid* boidsArrayHost;



/*
Cell class to represent a subdivision of the simulation space containing a list of pointers to boids
*/
struct Cell {
    std::list<Boid*> boids;
};



/*
Saves 
*/
void save(FILE *fptr, int numBoids, int frameNumber) {
    // Write vector array to file
    fprintf(fptr, "Frame %d\n", frameNumber);
    for(int i = 0; i < numBoids; i++) {
        fprintf(fptr, "%f %f %f %f\n", boidsArrayHost[i].px, boidsArrayHost[i].py, boidsArrayHost[i].vx, boidsArrayHost[i].vy);
    }    
}



// Sourced from https://stackoverflow.com/questions/686353/random-float-number-generation
float randFloat(float min, float max) {
    return min + static_cast <float> (rand()) /( static_cast <float> (RAND_MAX/(max-min)));
}



/*
Magnitude of a vector
*/
__device__ float mag(float x, float y) {
    return sqrt(x * x + y * y);
}



/*
Square magnitude of a vector
*/
__device__ float sqrMag(float x, float y) {
    return x * x + y * y;
}



/*
Vector dot product
*/
__device__ float dot(float x1, float y1, float x2, float y2) {
    return x1 * x2 + y1 * y2;
}



/*
Gets the orthogonal vector using pass-by-reference
*/
__device__ void getOrthogonal(float &orthogonalVector_x, float &orthogonalVector_y, 
        float diffVector_x, float diffVector_y, 
        float formationDir_x, float formationDir_y) {
    //TODO expand and simplify    
    // Determines whether formation is to the left or right of boid using cross product
    // And constructs formation vector accordingly by rotating formationDir
    float formationVector_x, formationVector_y;
    float c = cos(formationAngle);
    float s = sin(formationAngle);
    if (formationDir_x * diffVector_y - formationDir_y * diffVector_x > 0) { //If boid is to the left of formation
        formationVector_x = formationDir_x * c - formationDir_y * s;
        formationVector_y = formationDir_x * s - formationDir_y * c;
    }
    else { //If boid is to the right of formation
        formationVector_x = formationDir_x * c + formationDir_y * s;
        formationVector_y = formationDir_y * c - formationDir_x * s;
    }

    // Check that formation is ahead of boid not behind (the dot product is more than 90 degrees)
    // This is to stop the leaders from trying to fall in line behind
    if (dot(formationDir_x, formationDir_y, diffVector_x, diffVector_y) < 0) {
        // Get at the vector orthogonal to the formationVector and move in that direction
        float sqrM = sqrMag(formationVector_x, formationVector_y);
        float val = dot(diffVector_x, diffVector_y, formationVector_x, formationVector_y) / sqrM;
        orthogonalVector_x = val * formationVector_x - diffVector_x;
        orthogonalVector_y = val * formationVector_y - diffVector_y;
    }
}



//Using shared memory
__global__ void updateBoidsKernel_GPU(int N, Boid* in, Boid* out)
{
    //This boid's index
    int thisIndex = blockIdx.x * blockDim.x + threadIdx.x;

    if (thisIndex < N) { //Check out of bounds
        float avoidVector_x = 0, avoidVector_y = 0;
        float formationDir_x = 0, formationDir_y = 0;
        float formationPos_x = 0, formationPos_y = 0;
        int neighboringBoids = 0;

        //For some reason Boid& o = in[otherIndex]; seems to make it faster but making this line a reference makes it slower
        Boid b = in[thisIndex];

        for (int otherIndex = 0; otherIndex < N; otherIndex++) {
            if (otherIndex == thisIndex) continue; //Ignore itself

            Boid& o = in[otherIndex];
        
            // Get the distance between this boid and other boid
            float sqrDist = sqrMag(b.px - o.px, b.py - o.py);
            if (sqrDist < sqrAvoidRange) { // If the distance is less than protected range
                //Divide by the square of distance to make avoidance exponential and smoother
                avoidVector_x += (b.px - o.px) / sqrDist;
                avoidVector_y += (b.py - o.py) / sqrDist;
            }
            if (sqrDist < sqrVisualRange) { // If the distance is less than visual range
                formationDir_x += o.vx;
                formationDir_y += o.vy;
                formationPos_x += o.px;
                formationPos_y += o.py;
                neighboringBoids++;
            }
        }

        //Make a copy for out
        Boid& bOut = out[thisIndex]; //Reference to out[thisIndex]
        bOut = b;

        // Separation - move away from nearby boids
        bOut.vx += avoidVector_x * avoidFactor;
        bOut.vy += avoidVector_y * avoidFactor;

        if (neighboringBoids > 0) { //If there were any boids in visual range
            // Get mean formation direction and position
            formationDir_x /= neighboringBoids;
            formationDir_y /= neighboringBoids;
            formationPos_x /= neighboringBoids;
            formationPos_y /= neighboringBoids;  

            // Alignment - match the mean velocity of all boids in visual range
            bOut.vx += (formationDir_x - bOut.vx) * matchingFactor;
            bOut.vy += (formationDir_y - bOut.vy) * matchingFactor;

            // Flocking
            // Represents a vector pointed dowards this boid from the centre of mass
            float diffVector_x = bOut.px - formationPos_x;
            float diffVector_y = bOut.py - formationPos_y;

            float orthogonalVector_x = 0, orthogonalVector_y = 0;
            getOrthogonal(orthogonalVector_x, orthogonalVector_y, 
                    diffVector_x, diffVector_y,
                    formationDir_x, formationDir_y);

            bOut.vx += orthogonalVector_x * cohesionFactor;
            bOut.vy += orthogonalVector_y * cohesionFactor;            
        }



        // Avoid edges
        if (bOut.px < leftMargin) {
            bOut.vx += turnFactor;
        }
        else if (bOut.px > rightMargin) {
            bOut.vx -= turnFactor;
        }
        if (bOut.py < bottomMargin) {
            bOut.vy += turnFactor;
        }
        else if (bOut.py > topMargin) {
            bOut.vy -= turnFactor;
        }
        //---------------------------------------



        // Impose speed limit on boid
        float speed = mag(bOut.vx, bOut.vy);
        if (speed > maxSpeed) {
            bOut.vx *= maxSpeed / speed;
            bOut.vy *= maxSpeed / speed;
        }
        else if (speed < minSpeed) {
            bOut.vx *= minSpeed / speed;
            bOut.vy *= minSpeed / speed;
        }
        //else if (speed == 0) {} // TODO
        //---------------------------------------

        
        
        // Update boid position
        bOut.px += bOut.vx;
        bOut.py += bOut.vy;
        //---------------------------------------
    }
}



__host__ float updateBoids_GPU(int N)
{
    size_t size = N * sizeof(Boid);

    //Allocate memory on the device
    Boid *deviceIn;
    Boid *deviceOut;
    hipMalloc(&deviceIn, size);
    hipMalloc(&deviceOut, size);

    //Copy memory from host to device
    hipMemcpy(deviceIn, boidsArrayHost, size, hipMemcpyHostToDevice);

    //Specify blocks and threads, using 1D threads and blocks since our data is 1D
    int threadsPerBlock = BLOCKSIZE;
    int numBlocks = (N + threadsPerBlock - 1) / threadsPerBlock; //Int division of N / threadsPerBlock

    hipEvent_t startGPU;
    hipEvent_t stopGPU;
    hipEventCreate(&startGPU);
    hipEventCreate(&stopGPU);
    //Start Timer
    hipEventRecord(startGPU);
    
    //Run
    updateBoidsKernel_GPU<<<numBlocks, threadsPerBlock>>>(N, deviceIn, deviceOut);

    //Mark end of kernel execution
    hipEventRecord(stopGPU);
    //Wait for results to be available
    hipEventSynchronize(stopGPU);   
    //Measure the time the kernel took to execute
    float kernelTime;
    hipEventElapsedTime(&kernelTime, startGPU, stopGPU);

    //Wait for the kernel to synchronise
    hipDeviceSynchronize();

    //Copy memory from device to host
    hipMemcpy(boidsArrayHost, deviceOut, size, hipMemcpyDeviceToHost);

    //Clean up
    hipFree(deviceIn);
    hipFree(deviceOut);

    return kernelTime * 1000;
}



void init(int numBoids)
{
    boidsArrayHost = (Boid*)malloc(numBoids * sizeof(Boid));

    // Initialise array of boids and assign them to cells
    for(int i = 0; i < numBoids; i++) {
        float px = randFloat(0, xSize);
        float py = randFloat(0, ySize);
        // Random normalised direction
        float randTheta = randFloat(0, 2 * PI);
        float vx = cos(randTheta) * 1.0;
        float vy = sin(randTheta) * 1.0;
        boidsArrayHost[i] = Boid(px, py, vx, vy);
    }
}


/*
int main()
{
    int numBoids = 1000;
    int numFrames = 300;


    // Create a file and open it for writing
    FILE *fptr;
    fptr = fopen(filepath, "w");
    if (fptr == NULL) {
        printf("%s", "Error opening file");
        return 1;
    }


    //Variables for main
    setVars(512, 512, 64, 0.2, 8, 0.15, 20, 0.05, 0.2, 1, 2, 0.7 * PI);

    init(numBoids);

    save(fptr, numBoids, 0);



    // Update boids
    for (int frame = 1; frame < numFrames; frame++) {
        updateBoids_GPU(numBoids);
        
        save(fptr, numBoids, frame);
    }


    
    // Close the file
    fclose(fptr);



    return 0;
}
*/