
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdarg.h>

const char *filepath = "TestOutput.txt";

int main()
{
    // Create a file and open it for writing
    FILE *fptr;
    fptr = fopen(filepath, "w");
    if (fptr == NULL) {
        printf("%s", "Error opening file");
        return 1;
    }

    fprintf(fptr, "Hello, world\n");
}